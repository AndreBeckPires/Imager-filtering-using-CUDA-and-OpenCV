#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "opencv2/imgproc/imgproc.hpp"
#include <opencv2/highgui.hpp>
#include <iostream>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>



#define BLOCK_SIZE      16
#define FILTER_WIDTH    3     
#define FILTER_HEIGHT   3      

using namespace std;
using namespace cv;

__global__ void blurFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float kernel[FILTER_WIDTH][FILTER_HEIGHT] = { 0.0, 0.2, 0.0, 0.2, 0.2, 0.2, 0.0, 0.2, 0.0 };

	if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
	{
		for (int c = 0; c < channel; c++)
		{
		
			float sum = 0;
		
			for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
				for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
					float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
					sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
				}
			}
			dstImage[(y * width + x) * channel + c] = sum;
		}
	}
}

__global__ void embossFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float kernel[FILTER_WIDTH][FILTER_HEIGHT] = { -1, -1, 0, -1, 0, 1, 0.0, 1, 1 };
	
	if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
	{
		for (int c = 0; c < channel; c++)
		{
		
			float sum = 0;
	
			for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
				for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
					float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
					sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
				}
			}
			dstImage[(y * width + x) * channel + c] = sum + 128;
		}
	}
}


__global__ void sharpenFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	float kernel[FILTER_WIDTH][FILTER_HEIGHT] = { -1, -1, -1, -1, 9, -1, -1, -1, -1 };
	
	if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
	{
		for (int c = 0; c < channel; c++)
		{
	
			float sum = 0;
			
			for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
				for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
					float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
					sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
				}
			}
			dstImage[(y * width + x) * channel + c] = sum;
		}
	}
}

extern "C" void Filter_wrapper_blur(const Mat& input, Mat& output)
{


	
	int channel = input.step / input.cols;

   //seta o numero de bytes usando o cols e rows
	const int inputSize = input.cols * input.rows * channel;
	const int outputSize = output.cols * output.rows * channel;
	unsigned char* d_input, * d_output;

	//aloca memoria na grpu
	hipMalloc<unsigned char>(&d_input, inputSize);
	hipMalloc<unsigned char>(&d_output, outputSize);

	// passa a memoria pra gpu
	hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);

	//setta o grid da imegem
	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

	
	const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);



	// roda o filtro
	blurFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, channel);


 // memoria de volta pro host
	hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

	//limpa
	hipFree(d_input);
	hipFree(d_output);

}
extern "C" void Filter_wrapper_sharpen(const Mat& input, Mat& output)
{


	
	int channel = input.step / input.cols;

	
	const int inputSize = input.cols * input.rows * channel;
	const int outputSize = output.cols * output.rows * channel;
	unsigned char* d_input, * d_output;

	hipMalloc<unsigned char>(&d_input, inputSize);
	hipMalloc<unsigned char>(&d_output, outputSize);


	hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);


	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

	const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);



 
	sharpenFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, channel);




	hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);

}

extern "C" void Filter_wrapper_emboss(const Mat& input, Mat& output)
{


	int channel = input.step / input.cols;


	const int inputSize = input.cols * input.rows * channel;
	const int outputSize = output.cols * output.rows * channel;
	unsigned char* d_input, * d_output;


	hipMalloc<unsigned char>(&d_input, inputSize);
	hipMalloc<unsigned char>(&d_output, outputSize);


	hipMemcpy(d_input, input.ptr(), inputSize, hipMemcpyHostToDevice);


	const dim3 block(BLOCK_SIZE, BLOCK_SIZE);

	
	const dim3 grid((output.cols + block.x - 1) / block.x, (output.rows + block.y - 1) / block.y);



 
	embossFilter << <grid, block >> > (d_input, d_output, output.cols, output.rows, channel);




	hipMemcpy(output.ptr(), d_output, outputSize, hipMemcpyDeviceToHost);


	hipFree(d_input);
	hipFree(d_output);

}



